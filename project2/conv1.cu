#include "hip/hip_runtime.h"
#include <string>
#include <cstdlib>
#include <iostream>
#include <chrono>
#include <cassert>
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "../cuda_common.h"


using namespace std;


#ifdef CONV
  #define CONV_Ny 224
  #define CONV_Nx 224
  #define CONV_Ni 64
  #define CONV_Nn 64
#else
  #define CONV_Ny 14
  #define CONV_Nx 14
  #define CONV_Ni 512
  #define CONV_Nn 512
#endif

#define CONV_Ky 3
#define CONV_Kx 3
#define CONV_Sy 1
#define CONV_Sx 1

//Tiling Sizes
#define CONV_Tnn 32
#define CONV_Tn  16
#define CONV_Ti  16

#define CONV_Ty  8
#define CONV_Tx  8

#define CONV_NYPAD (CONV_Ny) // #define CONV_NYPAD (CONV_Ny+CONV_Ky)
#define CONV_NXPAD (CONV_Nx) // #define CONV_NXPAD (CONV_Nx+CONV_Kx)

#define CONV_NYSCL ((CONV_Ny - CONV_Ky + 1)/CONV_Sy) // #define CONV_NYSCL (CONV_Ny/CONV_Sy)
#define CONV_NXSCL ((CONV_Nx - CONV_Kx + 1)/CONV_Sx) // #define CONV_NXSCL (CONV_Nx/CONV_Sx)
                                                                                                                                   
#define CONV_FILTER_SIZE (CONV_Ky*CONV_Kx*CONV_Nn*CONV_Ni)
#define CONV_INPUT_SIZE (CONV_NYPAD*CONV_NXPAD*CONV_Ni)
#define CONV_OUTPUT_SIZE (CONV_NYSCL*CONV_NXSCL*CONV_Nn)

#define CONV_THREADS 1024
#define CONV_BLOCKS 500

using VTYPE = float;

#define OUTPUT_ADDR(ny, nx, nn) (CONV_Nx*CONV_Nn*(ny) + CONV_Nn*(nx) + (nn))
#define INPUT_ADDR(ni, ny, nx) (CONV_Ny*CONV_Nx*(ni) + CONV_Nx*(ny) + (nx))
#define KERNEL_ADDR(ni, nn, ky, kx) (CONV_Nn * CONV_Ky*CONV_Kx*(ni) + CONV_Ky*CONV_Kx*(nn) + CONV_Kx*(ky) + (kx))

bool is_gpu_cpu_arr_equal(VTYPE *output, VTYPE *cuOutput, int outputLen) {
    for(int i=0; i<outputLen; i++) {
        float diff = abs(output[i] - cuOutput[i])/(abs(cuOutput[i]) + 0.0001);
        if(diff > 0.05) {
            std::cout << "  Output: " << output[i] << std::endl;
            std::cout << "  cuOutput: " << cuOutput[i] << std::endl;
            std::cout << "  Diff: " << diff << std::endl;
            return false;
        }
    }
    return true;
}


/*
 * Run convolution using cuDNN
 * @param input: input tensor NCHW (# outputs, # input channels, height, width)
 * @param kernels: kernel tensor NCHW (# outputs, # input channels, height, width)
 * @param output: output tensor NCHW (# outputs, # input channels, height, width). 
 *  Output is assumed to be preallocated and is overwritten.
 * @return: void
 */
void runCUDNNConv(VTYPE *input, VTYPE *kernels, VTYPE *output)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnTensorDescriptor_t inputDesc, outputDesc;
    hipdnnFilterDescriptor_t kernelDesc;
    hipdnnConvolutionDescriptor_t convDesc;

    hipdnnCreateTensorDescriptor(&inputDesc);
    hipdnnCreateTensorDescriptor(&outputDesc);
    hipdnnCreateFilterDescriptor(&kernelDesc);
    hipdnnCreateConvolutionDescriptor(&convDesc);

    hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, CONV_Ni, CONV_Ny, CONV_Nx);
    hipdnnSetFilter4dDescriptor(kernelDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, CONV_Nn, CONV_Ni, CONV_Ky, CONV_Kx);
    hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, CONV_Nn, CONV_NYSCL, CONV_NXSCL);
    hipdnnSetConvolution2dDescriptor(convDesc, 0, 0, 1, 1, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

    VTYPE *cuInput, *cuKernels, *cuOutput;
    hipMalloc(&cuInput, CONV_INPUT_SIZE*sizeof(VTYPE));
    hipMalloc(&cuKernels, CONV_FILTER_SIZE*sizeof(VTYPE));
    hipMalloc(&cuOutput, CONV_OUTPUT_SIZE*sizeof(VTYPE));
    hipMemcpy(cuInput, input, CONV_INPUT_SIZE*sizeof(VTYPE), hipMemcpyHostToDevice);
    hipMemcpy(cuKernels, kernels, CONV_FILTER_SIZE*sizeof(VTYPE), hipMemcpyHostToDevice);
    float alpha = 1.0f, beta = 0.0f;
    hipEventRecord(start);
    hipdnnConvolutionForward(cudnn, &alpha, inputDesc, cuInput, kernelDesc, cuKernels, convDesc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, NULL, 0, &beta, outputDesc, cuOutput);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipMemcpy(output, cuOutput, CONV_OUTPUT_SIZE*sizeof(VTYPE), hipMemcpyDeviceToHost);
    // hipMemcpy(output, cuOutput, Nn*outNx*outNy*sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(cuInput);
    hipFree(cuKernels);
    hipFree(cuOutput);
    
    float cudnnTime;
    hipEventElapsedTime(&cudnnTime, start, stop);
    std::cout << "CUDNN Conv Time: " << cudnnTime <<"ms" << std::endl;
}


// Base CPU Version (no optimizations)
void convolution_layer_base(VTYPE *kernel, VTYPE *input, VTYPE *output) {
    for (int ny = 0; ny + CONV_Ky < CONV_Ny; ny += CONV_Sy) {
        for (int nx = 0; nx + CONV_Kx < CONV_Nx; nx += CONV_Sx) {
            for (int ky = 0; ky < CONV_Ky; ky++) {
                for (int kx = 0; kx < CONV_Kx; kx++) {
                    for (int ni = 0; ni < CONV_Ni; ni++) {
                        for (int nn = 0; nn < CONV_Nn; nn++) {
                          // Assumes output has already been pre-zero'd out.
                          output[OUTPUT_ADDR(ny, nx, nn)] += input[INPUT_ADDR(ni, ny + ky, nx + kx)] * kernel[KERNEL_ADDR(ni, nn, ky, kx)];
                        }
                    }
                }
            }
        }
    }
}

// Conv Layer DianNao implementation.
void convolution_layer(VTYPE *kernel, VTYPE *input, VTYPE *output) {
  VTYPE sum[CONV_Nn]={0};

  // — Original code — (excluding nn, ii loops)
  int yout = 0;
  for (int y = 0; y + CONV_Ky < CONV_Ny; y += CONV_Sy) { // tiling for y;
    int xout = 0;
    for (int x = 0; x + CONV_Kx < CONV_Nx; x += CONV_Sx) { // tiling for x;
      for (int nn = 0; nn < CONV_Nn; nn += CONV_Tn) {
        for (int n = nn; n < nn + CONV_Tn; n++) {
          sum[n]=0;
        }

        // sliding window;
        for (int ky = 0; ky < CONV_Ky; ky++)
          for (int kx = 0; kx < CONV_Kx; kx++)
            for (int n = nn; n < nn + CONV_Tn; n++)
              for (int i = 0; i < CONV_Ni; i++) {
                VTYPE sv = kernel[KERNEL_ADDR(i, n, ky, kx)]; // VTYPE sv = synapse[ky][kx][n][i];
                VTYPE nv = input[INPUT_ADDR(i, ky + y, kx + x)]; // neuron_i[ky + y][kx + x][i];
                sum[n]+=sv*nv;
              }
        for (int n = nn; n < nn + CONV_Tn; n++) {
            output[OUTPUT_ADDR(yout, xout, n)] = sum[n]; //   output[yout][xout][n] = sum[n];
        }
      }
      xout++; 
    }
    yout++;
  }
}

#define OUTPUT_ADDR(nn, ny, nx) (CONV_NXSCL*CONV_Nn*(ny) + CONV_Nn*(nx) + (nn))
#define INPUT_ADDR(ni, ny, nx) (CONV_Ny*CONV_Nx*(ni) + CONV_Nx*(ny) + (nx))
#define KERNEL_ADDR(ni, nn, ky, kx) (CONV_Nn * CONV_Ky*CONV_Kx*(ni) + CONV_Ky*CONV_Kx*(nn) + CONV_Kx*(ky) + (kx))
// const int X_DIM = 16;
// const int Y_DIM = 8;
// const int Z_DIM = 8;
// dim3 gridDim(CONV_NXSCL/X_DIM, CONV_NYSCL/Y_DIM, CONV_Nn/Z_DIM);
// dim3 blockDim(X_DIM, Y_DIM, Z_DIM);

__global__ void convolution_layer_parallelized_gpu(VTYPE *synapse_2, VTYPE *neuron_i_2, VTYPE *neuron_n) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    VTYPE sum = 0;

    // for (int ni = 0; ni < CONV_Ni; ni++) {
    //     sum +=           neuron_i_2[INPUT_ADDR(ni, y+0, x+0)] * synapse_2[KERNEL_ADDR(ni, z, 0, 0)]
    //                     + neuron_i_2[INPUT_ADDR(ni, y+0, x+1)] * synapse_2[KERNEL_ADDR(ni, z, 0, 1)]
    //                     + neuron_i_2[INPUT_ADDR(ni, y+0, x+2)] * synapse_2[KERNEL_ADDR(ni, z, 0, 2)]
    //                     + neuron_i_2[INPUT_ADDR(ni, y+1, x+0)] * synapse_2[KERNEL_ADDR(ni, z, 1, 0)]
    //                     + neuron_i_2[INPUT_ADDR(ni, y+1, x+1)] * synapse_2[KERNEL_ADDR(ni, z, 1, 1)]
    //                     + neuron_i_2[INPUT_ADDR(ni, y+1, x+2)] * synapse_2[KERNEL_ADDR(ni, z, 1, 2)]
    //                     + neuron_i_2[INPUT_ADDR(ni, y+2, x+0)] * synapse_2[KERNEL_ADDR(ni, z, 2, 0)]
    //                     + neuron_i_2[INPUT_ADDR(ni, y+2, x+1)] * synapse_2[KERNEL_ADDR(ni, z, 2, 1)]
    //                     + neuron_i_2[INPUT_ADDR(ni, y+2, x+2)] * synapse_2[KERNEL_ADDR(ni, z, 2, 2)];

    // }
    // neuron_n[OUTPUT_ADDR(z, y, x)] = 5;
    neuron_n[0] = 5;
}

__global__ void convolution_layer_tiled_gpu(VTYPE *kernel, VTYPE *input, VTYPE *output) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    const int blockDim_x = blockDim.x;;
    const int blockDim_y = blockDim.y;
    const int blockDim_z = blockDim.z;
    int iStride = blockDim.z;
    int tidz = threadIdx.z;
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;

    VTYPE sum = 0;

    __shared__ VTYPE cache[34 * 10 * 4];
    
    for (int nni = 0; nni < CONV_Ni; nni+=iStride) {
        cache[tidz * blockDim_x * blockDim_y + tidy * blockDim_x + tidx] = input[INPUT_ADDR(nni + tidz, y, x)];
        if (tidy >= blockDim_y - 2) {
            cache[tidz * blockDim_x * blockDim_y + (tidy + 2) * blockDim_x + tidx] = input[INPUT_ADDR(nni + tidz, y + 2, x)];
        }
        if (tidx >= blockDim_x - 2) {
            cache[tidz * blockDim_x * blockDim_y + tidy * blockDim_x + tidx + 2] = input[INPUT_ADDR(nni + tidz, y, x + 2)];
        }
        if (tidx >= blockDim_x - 2 && tidy >= blockDim_y - 2) {
            cache[tidz * blockDim_x * blockDim_y + (tidy + 2) * blockDim_x + tidx + 2] = input[INPUT_ADDR(nni + tidz, y + 2, x + 2)];
        }
        __syncthreads();

        for (int ni = 0; ni < iStride; ni++) {
            sum += cache[ni * blockDim_x * blockDim_y + tidy * blockDim_x + tidx] * kernel[KERNEL_ADDR(nni + ni, z, 0, 0)]
                + cache[ni * blockDim_x * blockDim_y + tidy * blockDim_x + tidx + 1] * kernel[KERNEL_ADDR(nni + ni, z, 0, 1)]
                + cache[ni * blockDim_x * blockDim_y + tidy * blockDim_x + tidx + 2] * kernel[KERNEL_ADDR(nni + ni, z, 0, 2)]
                + cache[ni * blockDim_x * blockDim_y + (tidy + 1) * blockDim_x + tidx] * kernel[KERNEL_ADDR(nni + ni, z, 1, 0)]
                + cache[ni * blockDim_x * blockDim_y + (tidy + 1) * blockDim_x + tidx + 1] * kernel[KERNEL_ADDR(nni + ni, z, 1, 1)]
                + cache[ni * blockDim_x * blockDim_y + (tidy + 1) * blockDim_x + tidx + 2] * kernel[KERNEL_ADDR(nni + ni, z, 1, 2)]
                + cache[ni * blockDim_x * blockDim_y + (tidy + 2) * blockDim_x + tidx] * kernel[KERNEL_ADDR(nni + ni, z, 2, 0)]
                + cache[ni * blockDim_x * blockDim_y + (tidy + 2) * blockDim_x + tidx + 1] * kernel[KERNEL_ADDR(nni + ni, z, 2, 1)]
                + cache[ni * blockDim_x * blockDim_y + (tidy + 2) * blockDim_x + tidx + 2] * kernel[KERNEL_ADDR(nni + ni, z, 2, 2)];
        }
        __syncthreads();
    }
    output[OUTPUT_ADDR(z, y, x)] = sum;
}

int main(const int argc, const char** argv) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    unsigned int threads = prop.maxThreadsPerBlock;
    unsigned int max_blocks_x = prop.maxGridSize[0];
    unsigned int max_blocks_y = prop.maxGridSize[1];
    unsigned int max_blocks_z = prop.maxGridSize[2];
    std::cout << "threads: " << threads << std::endl;
    std::cout << "max_blocks x: " << max_blocks_x << std::endl;
    std::cout << "max_blocks y: " << max_blocks_y << std::endl;
    std::cout << "max_blocks z: " << max_blocks_z << std::endl;

    VTYPE *kernel =  (VTYPE*)malloc(sizeof(VTYPE) * CONV_FILTER_SIZE);
    VTYPE *input = (VTYPE*)malloc(sizeof(VTYPE) * CONV_INPUT_SIZE);
    VTYPE *output =   (VTYPE*)malloc(sizeof(VTYPE) * CONV_OUTPUT_SIZE);
    VTYPE *output_validation = (VTYPE*)malloc(sizeof(VTYPE) * CONV_OUTPUT_SIZE);
    for(int i=0; i<CONV_FILTER_SIZE; i++)
        kernel[i] = (float)rand() / (float)RAND_MAX;
    for(int i=0; i<CONV_INPUT_SIZE; i++)
        input[i] = (float)rand() / (float)RAND_MAX;
    for(int i=0; i<CONV_OUTPUT_SIZE; i++) {
        output[i] = 0.0f;
        output_validation[i] = 0.0f;
    }
    const int X_DIM = 32;
    const int Y_DIM = 8;
    const int Z_DIM = 4;
    dim3 gridDim(CONV_NXSCL/X_DIM, CONV_NYSCL/Y_DIM, CONV_Nn/Z_DIM); //222/16=13, 222/8=28, 64/8=8, 13*28*8=2912
    dim3 blockDim(X_DIM, Y_DIM, Z_DIM); // 16*8*8 = 1024

    VTYPE *cuInput, *cuKernels, *cuOutput;
    CHECK_CUDA_ERROR(hipMalloc(&cuInput, CONV_INPUT_SIZE*sizeof(VTYPE)));
    CHECK_CUDA_ERROR(hipMalloc(&cuKernels, CONV_FILTER_SIZE*sizeof(VTYPE)));
    CHECK_CUDA_ERROR(hipMalloc(&cuOutput, CONV_OUTPUT_SIZE*sizeof(VTYPE)));
    CHECK_CUDA_ERROR(hipMemcpy(cuInput, input, CONV_INPUT_SIZE*sizeof(VTYPE), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(cuKernels, kernel, CONV_FILTER_SIZE*sizeof(VTYPE), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    convolution_layer_parallelized_gpu<<<gridDim, blockDim>>>(cuKernels, cuInput, cuOutput);
    // convolution_layer_tiled_gpu<<<gridDim, blockDim>>>(cuKernels, cuInput, cuOutput);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "GPU time: " << milliseconds << " ms" << std::endl;

    // CHECK_CUDA_ERROR(hipMemcpy(output, cuOutput, CONV_OUTPUT_SIZE*sizeof(VTYPE), hipMemcpyDeviceToHost));
    // std::cout << "output random indices " << output[0] << " " << output[13] << " " << output[23] << std::endl;
    
    runCUDNNConv(input, kernel, output_validation);
    // // // convolution_layer_base(kernel, input, output_validation);
    // // assert(is_gpu_cpu_arr_equal(output_validation, output, CONV_OUTPUT_SIZE));

    // CHECK_CUDA_ERROR(hipFree(cuOutput));
    // CHECK_CUDA_ERROR(hipFree(cuInput));
    // CHECK_CUDA_ERROR(hipFree(cuKernels));

    // free(kernel);
    // free(input);
    // free(output);
    // free(output_validation);

    // CHECK_LAST_CUDA_ERROR();
    // hipDeviceReset();
}


