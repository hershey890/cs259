/*
 * Compile with: nvcc classifier.cu -o classifier -std=c++11 -lcublas
 * Performance (already optimal)
 * -----------------------------
 * Time(%)      Time     Calls       Avg       Min       Max  Name
 *   1.87%  3.3943ms         5  678.85us  672.35us  700.70us  mat_mult_gpu(float*, float*, float*)
 *   0.37%  673.63us         1  673.63us  673.63us  673.63us  void gemv2T_kernel_val<int, int, float, float, float, float, int=128, int=16, int=4, int=4, bool=0, bool=0, cublasGemvParam
 *
 * Resources
 * ---------
 * https://siboehm.com/articles/22/CUDA-MMM
 * https://docs.nvidia.com/deeplearning/performance/dl-performance-matrix-multiplication/index.html
 */
#include <cstdlib>
#include <iostream>
#include <chrono>
#include <cassert>
#include <typeinfo>
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>

#include "hipblas.h"


const int Ni = 25088;
const int Nn = 4096;
const int nIters = 5; // # of times to average time calculation over

const int nBlocks = 500; // Titan V has 640 cores and 80 SM
const int nThreads = 1024; // divisible by 32, max 1024


/* https://leimao.github.io/blog/Proper-CUDA-Error-Checking/
 */
#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}


/* https://stackoverflow.com/questions/13041399/equivalent-of-cudageterrorstring-for-cublas
 */
#define CHECK_CUBLAS_ERROR(val) check_cublas((val), #val, __FILE__, __LINE__)
template <typename T>
void check_cublas(T err, const char* const func, const char* const file,
                  const int line)
{
    if (err != HIPBLAS_STATUS_SUCCESS) {
        std::string errorStr;
        switch(err)
        {
            case HIPBLAS_STATUS_SUCCESS: errorStr = "CUBLAS_STATUS_SUCCESS";
            case HIPBLAS_STATUS_NOT_INITIALIZED: errorStr = "CUBLAS_STATUS_NOT_INITIALIZED";
            case HIPBLAS_STATUS_ALLOC_FAILED: errorStr = "CUBLAS_STATUS_ALLOC_FAILED";
            case HIPBLAS_STATUS_INVALID_VALUE: errorStr = "CUBLAS_STATUS_INVALID_VALUE"; 
            case HIPBLAS_STATUS_ARCH_MISMATCH: errorStr = "CUBLAS_STATUS_ARCH_MISMATCH"; 
            case HIPBLAS_STATUS_MAPPING_ERROR: errorStr = "CUBLAS_STATUS_MAPPING_ERROR";
            case HIPBLAS_STATUS_EXECUTION_FAILED: errorStr = "CUBLAS_STATUS_EXECUTION_FAILED"; 
            case HIPBLAS_STATUS_INTERNAL_ERROR: errorStr = "CUBLAS_STATUS_INTERNAL_ERROR"; 
            // default: errorStr = "unknown error";
        }
        std::cerr << "CUBLAS Runtime Error at: " << file << ":" << line
                << std::endl;
        std::cerr << errorStr << " " << func << std::endl;

    }
}


#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
    err = hipGetLastError(); // done twice intenstinoally
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
    }
}


bool is_gpu_cpu_arr_equal(float *output, float *cuOutput, float outputLen) {
    for(int i=0; i<outputLen; i++) {
        float diff = abs(output[i] - cuOutput[i])/(abs(cuOutput[i]) + 0.0001);
        if(diff > 0.05) {
            std::cout << output[i] << " " << cuOutput[i] << " " << diff << std::endl;
            return false;
        }
    }
    return true;
}


void mat_mult_cpu(float *input, float *weights, float *output)
{
    for(int i=0; i<Nn; i++) {
        output[i] = 0;
        for(int j=0; j<Ni; j++)
            output[i] += weights[Ni*i + j]*input[j];
    }
}


// 111ms. for 32 blocks 1 thread, only works for 1 thread
__global__
void mat_mult_gpu_naive(int *input, int *weights, int *output) 
{
    int rowsPerBlock = Nn / gridDim.x;
    int iStart = blockIdx.x*rowsPerBlock;
    int jStart = threadIdx.x;
    int jStride = blockDim.x;
    for(int i=iStart; i<iStart+rowsPerBlock; i++) { //4096/32=128
        int sum = 0;
        for(int j=jStart; j<Ni; j += jStride) { // 25088/32=784
            sum += weights[Ni*i + j]*input[j];
        }
        output[i] += sum;
    }
}


// 650 us for 500 blocks and 1024 threads
__global__
void mat_mult_gpu(float *input, float *weights, float *output)
{    
    __shared__ int outputReduce[nThreads];

    int rowsPerBlock = (Nn + nBlocks - 1) / nBlocks;
    int iStart = blockIdx.x*rowsPerBlock;
    int tid = threadIdx.x;

    for(int i=iStart; i<iStart+rowsPerBlock && i<Nn; i++) {
        float sum = 0;
        for(int j=tid; j<Ni; j += nThreads)
            sum += weights[Ni*i + j]*input[j];
        outputReduce[tid] = sum;

        // Reduction - deals with 
        // https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
        __syncthreads();
        if(tid == 0) {
            sum = 0;
            for(int j=0; j<nThreads; j++)
                sum += outputReduce[j];
            output[i] = sum;
        }
    }
}


int main()
{
    // Create Weights, Inputs, and Outputs
    float *weights = (float*)malloc(Ni*Nn*sizeof(float));
    float *input   = (float*)malloc(Ni*sizeof(float));
    float *output  = (float*)malloc(Nn*sizeof(float));
    for(int i=0; i<Ni*Nn; i++)
        weights[i] = rand() % 10;
    for(int i=0; i<Ni; i++)
        input[i] = rand() % 10;

    // // Naive CPU Implementation
    double elapsedTime = 0;
    for(int i=0; i<nIters; i++) {
        auto time0 = std::chrono::steady_clock::now();
        mat_mult_cpu(input, weights, output);
        auto time1 = std::chrono::steady_clock::now();
        std::chrono::duration<double> elapsedSeconds = time1 - time0;
        elapsedTime += elapsedSeconds.count();
    }
    std::cout << "CPU Time:       " << elapsedTime/nIters << std::endl;

    // // GPU Setup
    float *cuWeights, *cuInput, *cuOutput;
    hipMalloc(&cuWeights, Ni*Nn*sizeof(float));
    hipMalloc(&cuInput,   Ni*sizeof(float));
    hipMalloc(&cuOutput,  Nn*sizeof(float));
    hipMemcpy(cuWeights, weights, Ni*Nn*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cuInput, input, Ni*sizeof(float), hipMemcpyHostToDevice);

    // GPU Implementation
    for(int i=0; i<nIters; i++) {
        // cudaMemset(cuOutput, 0, Nn*sizeof(int)); // only needed for the naive example
        auto time0 = std::chrono::steady_clock::now();
        mat_mult_gpu<<<nBlocks, nThreads>>>(cuInput, cuWeights, cuOutput);
        hipDeviceSynchronize();
        auto time1 = std::chrono::steady_clock::now();
        std::chrono::duration<double> elapsedSeconds = time1 - time0;
        elapsedTime += elapsedSeconds.count();
    }
    float *validationOutput = (float*)malloc(Nn*sizeof(float));
    hipMemcpy(validationOutput, cuOutput, Nn*sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Naive GPU Time: " << elapsedTime/nIters << std::endl;
    assert(is_gpu_cpu_arr_equal(output, validationOutput, Nn));

    /* CUBLAS Benchmark
     * Compares our kernel vs. CUBLAS performance
     * https://github.com/deeperlearning/professional-cuda-c-programming/blob/master/solutions/chapter08/cublas-matrix-matrix-async.cu
     */
    // Create the cuBLAS handle
    hipblasHandle_t handle = 0;
    hipblasCreate(&handle);

    // // Allocate device memory
    float *cublasInput, *cublasMatrix, *cublasOutput, *y;
    hipMalloc(&cublasInput,  sizeof(float) * Ni);
    hipMalloc(&cublasMatrix, sizeof(float) * Ni * Nn);
    hipMalloc(&cublasOutput, sizeof(float) * Nn);
    hipMalloc(&y, 1*sizeof(float));

    // // Transfer inputs to the device
    hipblasSetMatrix(Nn, Ni, sizeof(float), weights, Nn, cublasMatrix, Nn);
    hipblasSetVector(Ni, sizeof(float), input, Ni, cublasInput, Ni);

    // Execute Matrix Vector-Multiplication
    const float alpha = 1.0f;
    const float beta = 0;
    CHECK_CUBLAS_ERROR(hipblasSgemv(handle, HIPBLAS_OP_T, Ni, Nn, &alpha, cublasMatrix, Ni, cublasInput, 1, &beta, cublasOutput, 1));
    CHECK_CUBLAS_ERROR(hipblasGetVector(Nn, sizeof(float), cublasOutput, 1, validationOutput, 1));
    hipblasDestroy(handle);
    
    // Free Memory
    hipFree(cublasOutput);
    hipFree(cublasMatrix);
    hipFree(cublasInput);
    hipFree(cuOutput);
    hipFree(cuInput);
    hipFree(cuWeights);
    free(output);
    free(input);
    free(weights);
    hipDeviceReset();
    
    // Cuda Error Checking
    CHECK_LAST_CUDA_ERROR();

    return 0;
}