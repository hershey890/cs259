
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <hipblas.h>

void readPtsFile(std::string filename, float** src, float** dst, uint32_t* n_bytes) {
    std::ifstream stream(filename, std::ios::in | std::ios::binary);

    stream.read(reinterpret_cast<char *>(n_bytes), sizeof(uint32_t));

    *src = new (std::nothrow) float[*n_bytes / sizeof(float)];
    *dst = new (std::nothrow) float[*n_bytes / sizeof(float)];
    stream.read(reinterpret_cast<char *>(*src), *n_bytes);
    stream.read(reinterpret_cast<char *>(*dst), *n_bytes);

    stream.close();
}

// Source: https://github.com/pradyotsn/Matrix-Inverse-in-CUDA/blob/master/mat_inv.cu
void invert(float** src, float** dst, int n, int batchSize)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int *P, *INFO;

    hipMalloc(&P, n * batchSize * sizeof(int));
    hipMalloc(&INFO,  batchSize * sizeof(int));

    int lda = n;

    float **A = (float **)malloc(batchSize*sizeof(float *));
    float **A_d, *A_dflat;

    hipMalloc(&A_d,batchSize*sizeof(float *));
    hipMalloc(&A_dflat, n*n*batchSize*sizeof(float));

    A[0] = A_dflat;
    for (int i = 1; i < batchSize; i++)
        A[i] = A[i-1]+(n*n);

    hipMemcpy(A_d,A,batchSize*sizeof(float *),hipMemcpyHostToDevice);

    for (int i = 0; i < batchSize; i++)
        hipMemcpy(A_dflat+(i*n*n), src[i], n*n*sizeof(float), hipMemcpyHostToDevice);


    hipblasSgetrfBatched(handle,n,A_d,lda,P,INFO,batchSize);


    int INFOh[batchSize];
    hipMemcpy(INFOh,INFO,batchSize*sizeof(int),hipMemcpyDeviceToHost);

    for (int i = 0; i < batchSize; i++)
        if(INFOh[i]  != 0)
        {
            fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
            hipDeviceReset();
            exit(EXIT_FAILURE);
        }

    float **C = (float **)malloc(batchSize*sizeof(float *));
    float **C_d, *C_dflat;

    hipMalloc(&C_d,batchSize*sizeof(float *));
    hipMalloc(&C_dflat, n*n*batchSize*sizeof(float));
    C[0] = C_dflat;
    for (int i = 1; i < batchSize; i++)
        C[i] = C[i-1] + (n*n);
    hipMemcpy(C_d,C,batchSize*sizeof(float *),hipMemcpyHostToDevice);
    hipblasSgetriBatched(handle,n,(const float **)A_d,lda,P,C_d,lda,INFO,batchSize);

    hipMemcpy(INFOh,INFO,batchSize*sizeof(int),hipMemcpyDeviceToHost);

    for (int i = 0; i < batchSize; i++)
        if(INFOh[i] != 0)
        {
            fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", i);
            hipDeviceReset();
            exit(EXIT_FAILURE);
        }
    for (int i = 0; i < batchSize; i++)
        hipMemcpy(dst[i], C_dflat + (i*n*n), n*n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(A_d); hipFree(A_dflat); free(A);
    hipFree(C_d); hipFree(C_dflat); free(C);
    hipFree(P); hipFree(INFO); hipblasDestroy(handle);
}

void linearRegressorFit(float* X, float* y, float* params, uint32_t N) {
    // Pad X on the top with ones
    float *cublasXPadded;
    float *ones = new float[N];
    for (int i = 0; i < N; i++)
        ones[i] = 1.0f;
    hipMalloc(&cublasXPadded, sizeof(float) * N * 2);
    hipMemcpy(cublasXPadded, ones, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(cublasXPadded + N, X, sizeof(float) * N, hipMemcpyHostToDevice);
    delete[] ones;

//    std::cout << "y:";
//    for (int i = 0; i < N; i++) {
//        std::cout << ' ' << y[i];
//    }
//    std::cout << '\n';

//    float *padded = new float[N * 2];
//    cudaMemcpy(padded, cublasXPadded, sizeof(float) * N * 2, cudaMemcpyDeviceToHost);
//    std::cout << "Padded:";
//    for (int i = 0; i < N * 2; i++) {
//        std::cout << ' ' << padded[i];
//    }
//    std::cout << '\n';

    // Perform outer product of cublasMatMul1 = cublasXPadded.T x cublasXPadded
    float *cublasMatMul1;
    const float alpha = 1.0f; const float beta = 1.0f;
    hipMalloc(&cublasMatMul1, sizeof(float) * 4);
    hipMemset(cublasMatMul1, 0, sizeof(float) * 4);
    hipblasHandle_t handle = 0;
    hipblasCreate(&handle);
    hipblasSgemm(
            handle,
            HIPBLAS_OP_T, HIPBLAS_OP_N,
            2, 2, N,
            &alpha,
            cublasXPadded, N,
            cublasXPadded, N,
            &beta,
            cublasMatMul1, 2
    );

//    float* matMul1 = new float[4];
//    cudaMemcpy(matMul1, cublasMatMul1, sizeof(float) * 4, cudaMemcpyDeviceToHost);
//    std::cout << "matMul1:";
//    for (int i = 0; i < 4; i++) {
//        std::cout << ' ' << matMul1[i];
//    }
//    std::cout << '\n';

    // Invert X.T x X
    float **srcMatrix, **dstMatrix;
    srcMatrix = new float*[1];
    dstMatrix = new float*[1];
    srcMatrix[0] = new float[4];
    dstMatrix[0] = new float[4];
    hipMemcpy(srcMatrix[0], cublasMatMul1, sizeof(float) * 4, hipMemcpyDeviceToHost);
    invert(srcMatrix, dstMatrix, 2, 1);

//    std::cout << "Inverted matrix:";
//    for (int i = 0; i < 4; i++) {
//        std::cout << ' ' << dstMatrix[0][i];
//    }
//    std::cout << '\n';

    // Multiply inverse(X.T x X) by X.T
    float *cublasInverseMat, *cublasMatMul2;
    hipMalloc(&cublasInverseMat, sizeof(float) * 4);
    hipMalloc(&cublasMatMul2, sizeof(float) * N * 2);
    hipMemcpy(cublasInverseMat, dstMatrix[0], sizeof(float) * 4, hipMemcpyHostToDevice);
    hipMemset(cublasMatMul2, 0, sizeof(float) * N * 2);
    hipblasSgemm(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_T,
            2, N, 2,
            &alpha,
            cublasInverseMat, 2,
            cublasXPadded, N,
            &beta,
            cublasMatMul2, 2
    ); // Gets the transpose of the answer?

//    float* matMul2 = new float[N * 2];
//    cudaMemcpy(matMul2, cublasMatMul2, sizeof(float) * N * 2, cudaMemcpyDeviceToHost);
//    std::cout << "MatMul2:";
//    for (int i = 0; i < N * 2; i++) {
//        std::cout << ' ' << matMul2[i];
//    }
//    std::cout << '\n';

    // Multiply (inverse(X.T x X) by X.T) by y
    float *cublasY, *cublasMatMul3;
    hipMalloc(&cublasY, sizeof(float) * N);
    hipMalloc(&cublasMatMul3, sizeof(float) * 2);
    hipblasSetVector(N, sizeof(float), y, 1, cublasY, 1);
    hipMemset(cublasMatMul3, 0, sizeof(float) * 2);
    hipblasSgemv(
            handle, HIPBLAS_OP_N,
            2, N,
            &alpha,
            cublasMatMul2, 2,
            cublasY, 1,
            &beta,
            cublasMatMul3, 1
    );

//    float* matMul3 = new float[2];
//    cudaMemcpy(matMul3, cublasMatMul3, sizeof(float) * 2, cudaMemcpyDeviceToHost);
//    std::cout << "MatMul3:";
//    for (int i = 0; i < 2; i++) {
//        std::cout << ' ' << matMul3[i];
//    }
//    std::cout << '\n';

    hipMemcpy(params, cublasMatMul3, sizeof(float) * 2, hipMemcpyDeviceToHost);

    // Deletes
    delete srcMatrix[0];
    delete dstMatrix[0];
    delete srcMatrix;
    delete dstMatrix;

    // Cuda Frees
    hipFree(cublasXPadded);
    hipFree(cublasMatMul1);
    hipFree(cublasMatMul2);
    hipFree(cublasMatMul3);
    hipFree(cublasY);
    hipFree(cublasInverseMat);

    // Cublas Destroy Handle
    hipblasDestroy(handle);
}

void linearRegressorPredict(float* X, float* params, uint32_t N) {
    // Pad X on the top with ones
    float *cublasXPadded;
    float *ones = new float[N];
    for (int i = 0; i < N; i++)
        ones[i] = 1.0f;
    hipMalloc(&cublasXPadded, sizeof(float) * N * 2);
    hipMemcpy(cublasXPadded, ones, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(cublasXPadded + N, X, sizeof(float) * N, hipMemcpyHostToDevice);
    delete[] ones;

//    float *padded = new float[N * 2];
//    cudaMemcpy(padded, cublasXPadded, sizeof(float) * N * 2, cudaMemcpyDeviceToHost);
//    std::cout << "Padded:";
//    for (int i = 0; i < N * 2; i++) {
//        std::cout << ' ' << padded[i];
//    }
//    std::cout << '\n';
//
//    std::cout << "Params:";
//    for (int i = 0; i < 2; i++) {
//        std::cout << ' ' << params[i];
//    }
//    std::cout << '\n';

    // Perform the prediction calculation
    float *cublasParams, *cublasMatMul;
    const float alpha = 1.0f; const float beta = 1.0f;
    hipMalloc(&cublasParams, sizeof(float) * 2);
    hipMalloc(&cublasMatMul, sizeof(float) * N);
    hipMemcpy(cublasParams, params, sizeof(float) * 2, hipMemcpyHostToDevice);
    hipMemset(cublasMatMul, 0, sizeof(float) * N);
    hipblasHandle_t handle = 0;
    hipblasCreate(&handle);
    hipblasSgemm(
            handle,
            HIPBLAS_OP_N, HIPBLAS_OP_T,
            1, N, 2,
            &alpha,
            cublasParams, 1,
            cublasXPadded, N,
            &beta,
            cublasMatMul, 1
    );

//    float *matMul = new float[N];
//    cudaMemcpy(matMul, cublasMatMul, sizeof(float) * N, cudaMemcpyDeviceToHost);
//    std::cout << "MatMul:";
//    for (int i = 0; i < N; i++) {
//        std::cout << ' ' << matMul[i];
//    }
//    std::cout << '\n';

    // Cuda Frees
    hipFree(cublasXPadded);
    hipFree(cublasParams);
    hipFree(cublasMatMul);

    // Cublas Destroy Handle
    hipblasDestroy(handle);
}

int main()
{
    float *src, *dst;
    uint32_t n_bytes;
    readPtsFile("./data/src_dst_pts.bin", &src, &dst, &n_bytes);

//    float **inv_src, **inv_dst;
//    inv_src = new float*[1];
//    inv_dst = new float*[1];
//    inv_src[0] = new float[9];
//    inv_dst[0] = new float[9];
//    inv_src[0][0] = inv_src[0][4] = inv_src[0][8] = 2.0f;
//    invert(inv_src, inv_dst, 3, 1);

    float *X, *y, *params;
    X = new float[3];
    y = new float[3];
    params = new float[2];
    X[0] = 1; X[1] = 2; X[2] = 3;
    y[0] = 7; y[1] = 8; y[2] = 9;
    linearRegressorFit(X, y, params, 3);
    linearRegressorPredict(X, params, 3);
}
